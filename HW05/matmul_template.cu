#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include "cstdio"
#include <typeinfo>

// timer
hipEvent_t start;
hipEvent_t stop;

template<typename T>
__global__ void kernel_matmul(T *A, T *B, T *C, uint n, uint block_dim);

template<typename T>
__host__ void matmul(const T *A, const T *B, const T *C, uint n, uint block_dim);

namespace cuda{
  template<typename T>
  struct SharedMemory
  {
    __device__ T* getPointer(){
      return (T*)0;
    }
  };

  template<>
  struct SharedMemory <int>
  {
    __device__ int* getPointer(){extern __shared__ int s_int[]; return s_int;}
  };

  template<>
  struct SharedMemory <float>
  {
    __device__ float* getPointer(){extern __shared__ float s_float[]; return s_float;}
  };

  template<>
  struct SharedMemory <double>
  {
    __device__ double* getPointer(){extern __shared__ double s_double[]; return s_double;}
  };  
}


__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
                       unsigned int block_dim)
{
    matmul(A, B, C, n, block_dim);
}
__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
                       unsigned int block_dim)
{
    matmul(A, B, C, n, block_dim);
}
__host__ void matmul_3(const double *A, const double *B, double *C,
                       unsigned int n, unsigned int block_dim)
{
    matmul(A, B, C, n, block_dim);
}

template<typename T>
__global__ void kernel_matmul(T *A, T *B, T *C, uint n,
                              uint block_dim) {
  // allocate share memory
  cuda::SharedMemory<T> sharedMemory;
  T* shrMem = sharedMemory.getPointer();
  T *shrMemA = shrMem; // shrMem for matrix A
  T *shrMemB =
      shrMem + block_dim * block_dim + 1; // shrMem for matrix B
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
    }
    if (ny < n) {
      // put data into shrMem
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}

template<typename T>
__host__ void matmul(const T *A, const T *B, const T *C, uint n, uint block_dim){
    // allocate memory in GPU
    T *dA, *dB, *dC;
    hipMalloc((void **)&dA, n * n * sizeof(T));
    hipMalloc((void **)&dB, n * n * sizeof(T));
    hipMalloc((void **)&dC, n * n * sizeof(T));

    // copy data into device
    hipMemcpy(dA, A, n * n * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(T), hipMemcpyHostToDevice);
    hipMemset(dC, 0, n * n * sizeof(T));
    // kernel configuration
    dim3 DimBlock(block_dim, block_dim);
    dim3 DimGrid((n + block_dim - 1) / block_dim,
                (n + block_dim - 1) / block_dim);

    // timer
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // call the kernel function
    kernel_matmul<<<DimGrid, DimBlock, 2 * block_dim * block_dim * sizeof(T)>>>(
        dA, dB, dC, n, block_dim);
    
    // timer
    hipEventRecord(stop);
    hipEventSynchronize(stop); 

    // copy the results
    hipMemcpy((void *)C, (void *)dC, n * n * sizeof(T), hipMemcpyDeviceToHost);

    // print out results
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    char printfCmd[20];
    if (typeid(T) == typeid(int)) {
      strcpy(printfCmd, "%d\n%d\n%f\n");
    } else {
      strcpy(printfCmd, "%f\n%f\n%f\n");
    }
    printf(printfCmd, C[0], C[n * n - 1], ms);

    // deallocate
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    dA = nullptr;
    dB = nullptr;
    dC = nullptr;
}

// int main(){
//     uint N = 5;
//     uint block_dim = 2;

//     // allocate
//     int *A = new int[N*N];
//     int *B = new int[N*N];
//     int *C = new int[N*N]{0};
//     srand(time(NULL));
//     for(uint i=0; i<N*N; i++){
//         // A[i] = (double)rand()/((double)RAND_MAX);
//         // B[i] = (double)rand()/((double)RAND_MAX);
//         A[i] = i;
//         B[i] = N*N-1-i;
//     }

//     matmul_1(A,B,C,N,block_dim);
//     for(uint i=0; i<N*N; i++){
//         printf("%d\n", C[i]);
//     }
// }