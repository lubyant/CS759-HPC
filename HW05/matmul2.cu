#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <stdio.h>
typedef unsigned int uint;
// timer
hipEvent_t start;
hipEvent_t stop;

__global__ void kernel_matmul1(const int *A, const int *B, int *C, uint n, uint block_dim);
__global__ void kernel_matmul2(const float *A, const float *B, float *C, uint n,
                              uint block_dim);
__global__ void kernel_matmul3(const double *A, const double *B, double *C, uint n,
                              uint block_dim);

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
                       unsigned int block_dim) {

  // kernel configuration
  dim3 DimBlock(block_dim, block_dim);
  dim3 DimGrid((n + block_dim - 1) / block_dim,
               (n + block_dim - 1) / block_dim);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // call the kernel function
  kernel_matmul1<<<DimGrid, DimBlock, 2 * block_dim * block_dim * sizeof(int)>>>(
      A, B, C, n, block_dim);

  // timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);


  // print out results
//   float ms;
//   hipEventElapsedTime(&ms, start, stop);
//   printf("%d\n%d\n%f\n", C[0], C[n * n - 1], ms);


}
__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
                       unsigned int block_dim) {
  // kernel configuration
  dim3 DimBlock(block_dim, block_dim);
  dim3 DimGrid((n + block_dim - 1) / block_dim,
               (n + block_dim - 1) / block_dim);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // call the kernel function
  kernel_matmul2<<<DimGrid, DimBlock, 2 * block_dim * block_dim * sizeof(float)>>>(
      A, B, C, n, block_dim);

  // timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // print out results
//   float ms;
//   hipEventElapsedTime(&ms, start, stop);
//   printf("%f\n%f\n%f\n", C[0], C[n * n - 1], ms);

}
__host__ void matmul_3(const double *A, const double *B, double *C,
                       unsigned int n, unsigned int block_dim) {

  // kernel configuration
  dim3 DimBlock(block_dim, block_dim);
  dim3 DimGrid((n + block_dim - 1) / block_dim,
               (n + block_dim - 1) / block_dim);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // call the kernel function
  kernel_matmul3<<<DimGrid, DimBlock, 2 * block_dim * block_dim * sizeof(double)>>>(
      A, B, C, n, block_dim);

  // timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);


  // print out results
//   float ms;
//   hipEventElapsedTime(&ms, start, stop);
//   printf("%f\n%f\n%f\n", C[0], C[n * n - 1], ms);

}

__global__ void kernel_matmul1(const int *A, const int *B, int *C, uint n, uint block_dim) {
  // allocate share memory
  __shared__ extern int shrMemInt[];
//   __shared__ int shrMemInt[2048];
  int *shrMemA = shrMemInt;                             // shrMem for matrix A
  int *shrMemB = shrMemInt + block_dim * block_dim + 1; // shrMem for matrix B
  // memset(shrMem, 0, 2*block_dim*block_dim*sizeof(T));
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
      // shrMemB[threadIdx.x*block_dim+threadIdx.y] =
      // B[(k*block_dim+threadIdx.x)*n+threadIdx.y+blockIdx.y*block_dim];
    }
    if (ny < n) {
      // put data into shrMem
      // shrMemA[threadIdx.x*block_dim+threadIdx.y] =
      // A[k*block_dim+threadIdx.y+(blockIdx.x*block_dim+threadIdx.x)*n];
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}

__global__ void kernel_matmul2(const float *A, const float *B, float *C, uint n,
                              uint block_dim) {
  // allocate share memory
  __shared__ extern float shrMemFlt[];
//   __shared__ float shrMemFlt[2048];
  float *shrMemA = shrMemFlt;                             // shrMem for matrix A
  float *shrMemB = shrMemFlt + block_dim * block_dim + 1; // shrMem for matrix B
  // memset(shrMem, 0, 2*block_dim*block_dim*sizeof(T));
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
      // shrMemB[threadIdx.x*block_dim+threadIdx.y] =
      // B[(k*block_dim+threadIdx.x)*n+threadIdx.y+blockIdx.y*block_dim];
    }
    if (ny < n) {
      // put data into shrMem
      // shrMemA[threadIdx.x*block_dim+threadIdx.y] =
      // A[k*block_dim+threadIdx.y+(blockIdx.x*block_dim+threadIdx.x)*n];
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}

__global__ void kernel_matmul3(const double *A, const double *B, double *C, uint n,
                              uint block_dim) {
  // allocate share memory
  __shared__ extern double shrMemDouble[];
//   __shared__ double shrMemDouble[2048];
  double *shrMemA = shrMemDouble; // shrMem for matrix A
  double *shrMemB =
      shrMemDouble + block_dim * block_dim + 1; // shrMem for matrix B
  // memset(shrMem, 0, 2*block_dim*block_dim*sizeof(T));
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
      // shrMemB[threadIdx.x*block_dim+threadIdx.y] =
      // B[(k*block_dim+threadIdx.x)*n+threadIdx.y+blockIdx.y*block_dim];
    }
    if (ny < n) {
      // put data into shrMem
      // shrMemA[threadIdx.x*block_dim+threadIdx.y] =
      // A[k*block_dim+threadIdx.y+(blockIdx.x*block_dim+threadIdx.x)*n];
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}
