#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <stdio.h>
typedef unsigned int uint;
// timer
hipEvent_t start;
hipEvent_t stop;

__global__ void kernel_matmul1(int *A, int *B, int *C, uint n, uint block_dim);
__global__ void kernel_matmul2(float *A, float *B, float *C, uint n,
                              uint block_dim);
__global__ void kernel_matmul3(double *A, double *B, double *C, uint n,
                              uint block_dim);

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
                       unsigned int block_dim) {
  // allocate memory in GPU
  int *dA, *dB, *dC;
  hipMalloc((void **)&dA, n * n * sizeof(int));
  hipMalloc((void **)&dB, n * n * sizeof(int));
  hipMalloc((void **)&dC, n * n * sizeof(int));

  // copy data into device
  hipMemcpy(dA, A, n * n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dB, B, n * n * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(dC, 0, n * n * sizeof(int));
  // kernel configuration
  dim3 DimBlock(block_dim, block_dim);
  dim3 DimGrid((n + block_dim - 1) / block_dim,
               (n + block_dim - 1) / block_dim);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // call the kernel function
  kernel_matmul1<<<DimGrid, DimBlock>>>(
      dA, dB, dC, n, block_dim);

  // timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // copy the results
  hipMemcpy(C, dC, n * n * sizeof(int), hipMemcpyDeviceToHost);

  // print out results
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  printf("%d\n%d\n%f\n", C[0], C[n * n - 1], ms);

  // deallocate
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  dA = nullptr;
  dB = nullptr;
  dC = nullptr;
}
__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
                       unsigned int block_dim) {
  // allocate memory in GPU
  float *dA, *dB, *dC;
  hipMalloc((void **)&dA, n * n * sizeof(float));
  hipMalloc((void **)&dB, n * n * sizeof(float));
  hipMalloc((void **)&dC, n * n * sizeof(float));

  // copy data into device
  hipMemcpy(dA, A, n * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dB, B, n * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemset(dC, 0, n * n * sizeof(float));
  // kernel configuration
  dim3 DimBlock(block_dim, block_dim);
  dim3 DimGrid((n + block_dim - 1) / block_dim,
               (n + block_dim - 1) / block_dim);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // call the kernel function
  kernel_matmul2<<<DimGrid, DimBlock>>>(
      dA, dB, dC, n, block_dim);

  // timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // copy the results
  hipMemcpy(C, dC, n * n * sizeof(float), hipMemcpyDeviceToHost);

  // print out results
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  printf("%f\n%f\n%f\n", C[0], C[n * n - 1], ms);

  // deallocate
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  dA = nullptr;
  dB = nullptr;
  dC = nullptr;
}
__host__ void matmul_3(const double *A, const double *B, double *C,
                       unsigned int n, unsigned int block_dim) {
  // allocate memory in GPU
  double *dA, *dB, *dC;
  hipMalloc((void **)&dA, n * n * sizeof(double));
  hipMalloc((void **)&dB, n * n * sizeof(double));
  hipMalloc((void **)&dC, n * n * sizeof(double));

  // copy data into device
  hipMemcpy(dA, A, n * n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dB, B, n * n * sizeof(double), hipMemcpyHostToDevice);
  hipMemset(dC, 0, n * n * sizeof(double));
  // kernel configuration
  dim3 DimBlock(block_dim, block_dim);
  dim3 DimGrid((n + block_dim - 1) / block_dim,
               (n + block_dim - 1) / block_dim);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // call the kernel function
  kernel_matmul3<<<DimGrid, DimBlock>>>(
      dA, dB, dC, n, block_dim);

  // timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // copy the results
  hipMemcpy(C, dC, n * n * sizeof(double), hipMemcpyDeviceToHost);

  // print out results
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  printf("%f\n%f\n%f\n", C[0], C[n * n - 1], ms);

  // deallocate
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  dA = nullptr;
  dB = nullptr;
  dC = nullptr;
}

__global__ void kernel_matmul1(int *A, int *B, int *C, uint n, uint block_dim) {
  // allocate share memory
  // __shared__ extern int shrMemInt[];
  __shared__ int shrMemInt[2048];
  int *shrMemA = shrMemInt;                             // shrMem for matrix A
  int *shrMemB = shrMemInt + block_dim * block_dim + 1; // shrMem for matrix B
  // memset(shrMem, 0, 2*block_dim*block_dim*sizeof(T));
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
      // shrMemB[threadIdx.x*block_dim+threadIdx.y] =
      // B[(k*block_dim+threadIdx.x)*n+threadIdx.y+blockIdx.y*block_dim];
    }
    if (ny < n) {
      // put data into shrMem
      // shrMemA[threadIdx.x*block_dim+threadIdx.y] =
      // A[k*block_dim+threadIdx.y+(blockIdx.x*block_dim+threadIdx.x)*n];
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}

__global__ void kernel_matmul2(float *A, float *B, float *C, uint n,
                              uint block_dim) {
  // allocate share memory
  // __shared__ extern float shrMemFlt[];
  __shared__ float shrMemFlt[2048];
  float *shrMemA = shrMemFlt;                             // shrMem for matrix A
  float *shrMemB = shrMemFlt + block_dim * block_dim + 1; // shrMem for matrix B
  // memset(shrMem, 0, 2*block_dim*block_dim*sizeof(T));
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
      // shrMemB[threadIdx.x*block_dim+threadIdx.y] =
      // B[(k*block_dim+threadIdx.x)*n+threadIdx.y+blockIdx.y*block_dim];
    }
    if (ny < n) {
      // put data into shrMem
      // shrMemA[threadIdx.x*block_dim+threadIdx.y] =
      // A[k*block_dim+threadIdx.y+(blockIdx.x*block_dim+threadIdx.x)*n];
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}

__global__ void kernel_matmul3(double *A, double *B, double *C, uint n,
                              uint block_dim) {
  // allocate share memory
  // __shared__ extern double shrMemDouble[];
  __shared__ double shrMemDouble[2048];
  double *shrMemA = shrMemDouble; // shrMem for matrix A
  double *shrMemB =
      shrMemDouble + block_dim * block_dim + 1; // shrMem for matrix B
  // memset(shrMem, 0, 2*block_dim*block_dim*sizeof(T));
  /**
   * index conversion
   * grid (blockIndx.x,blockIndex.y)
   * block (threadIdx.x, threadIdx.y)
   * nx = blockIdx.x*block_dim + threadIdx.x
   * ny = blockIdx.y*block_dim + threadIdx.y
   */
  uint nx = blockIdx.x * block_dim + threadIdx.x;
  uint ny = blockIdx.y * block_dim + threadIdx.y;

  for (uint k = 0; k < gridDim.x; k++) {
    // protection
    if (nx < n) {
      // put data into shrMem
      shrMemA[threadIdx.x * block_dim + threadIdx.y] =
          A[k * block_dim + threadIdx.y +
            (blockIdx.x * block_dim + threadIdx.x) * n];
      // shrMemB[threadIdx.x*block_dim+threadIdx.y] =
      // B[(k*block_dim+threadIdx.x)*n+threadIdx.y+blockIdx.y*block_dim];
    }
    if (ny < n) {
      // put data into shrMem
      // shrMemA[threadIdx.x*block_dim+threadIdx.y] =
      // A[k*block_dim+threadIdx.y+(blockIdx.x*block_dim+threadIdx.x)*n];
      shrMemB[threadIdx.x * block_dim + threadIdx.y] =
          B[(k * block_dim + threadIdx.x) * n + threadIdx.y +
            blockIdx.y * block_dim];
    }
    __syncthreads();
    // protection
    if (nx < n && ny < n) {
      // calculate inner product
      for (uint i = 0; i < block_dim; i++) {
        C[nx * n + ny] += shrMemA[threadIdx.x * block_dim + i] *
                          shrMemB[block_dim * i + threadIdx.y];
        // if(threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
        // blockIdx.y == 2) printf("%d*%d, %d \n",
        // shrMemA[threadIdx.x*block_dim+i],shrMemB[threadIdx.x*block_dim+threadIdx.y+i*block_dim],
        // C[nx*n+ny]);
      }
    }
    __syncthreads();
  }
}
