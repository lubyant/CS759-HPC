#include "mmul.h"
#include <hip/hip_runtime.h>

/**
 * @brief matrix multiplication use cublas library
 *
 * @param handle cublasHandle
 * @param A matrix A
 * @param B matrix B
 * @param C matrix C
 * @param n size of square matrix
 */
void mmul(hipblasHandle_t handle, const float *A, const float *B, float *C,
          int n) {
  // parameter for the cublas
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  float alpha = 1, beta = 1;
  float *alpha_p = &alpha;
  float *beta_p = &beta;

  // call the cublas routine
  hipblasSgemm(handle, transa, transb, n, n, n, alpha_p, A, n, B, n, beta_p,
                 C, n);
}