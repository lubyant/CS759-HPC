#include "vscale.cuh"
#include <stdio.h>

int main(int argc, char *argv[]) {
  using namespace std;
  hipEvent_t start;
  hipEvent_t stop;
  int n = atoi(argv[1]);
  // int n = 4;

  // allocate the dynamic array a and b
  float *a = new float[n];
  float *b = new float[n];

  // random initialization
  for (int i = 0; i < n; i++) {
    a[i] = 20 * ((float)rand()) / ((float)RAND_MAX) - 10;
    b[i] = ((float)rand()) / ((float)RAND_MAX);
  }

  // allocate array da,db in cuda device and copy a,b into da,db
  float *da;
  hipMalloc((void **)&da, sizeof(float) * n);
  hipMemcpy(da, a, sizeof(float) * n, hipMemcpyHostToDevice);
  float *db;
  hipMalloc((void **)&db, sizeof(float) * n);
  hipMemcpy(db, b, sizeof(float) * n, hipMemcpyHostToDevice);

  // timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // allocate the threads
  int threadsPerBlock = 256; // number of threads each block
  int blocksPerGrid =
      (n + threadsPerBlock - 1) / threadsPerBlock; // number of blocks
  vscale<<<blocksPerGrid, threadsPerBlock>>>(da, db, (unsigned int)n);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // copy from device into host for print
  hipMemcpy(b, db, n * sizeof(float), hipMemcpyDeviceToHost);
  float ms;
  hipEventElapsedTime(&ms, start, stop);

  // print the anser
  printf("%f\n%f\n%f\n", ms, b[0], b[n - 1]);

  // deallocate
  delete[] a;
  delete[] b;
  hipFree(da);
  hipFree(db);

  return 0;
}